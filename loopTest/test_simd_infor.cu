
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 1024*1024

__global__ void compare(const unsigned int* a, const unsigned int* b, double *rd, unsigned int *c) //kernel func, run on GPU
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;;    // this thread handles the data at its thread id
    //if(tid >= N) return;
    double x = rd[tid];
    // SIMD Version
    //unsigned int condition = __vsetne4(a[tid], b[tid]);
    for(std::size_t i =0; i < 1e5 ; ++i)
    {
        if(__vsetne4(a[tid], b[tid])) {
            x = 4 * x * (x-1);
        } else {
            x = 3.8 * x * (x-1);
        }
    }
    rd[tid] = x;
}
 
int cpu_compare(unsigned int *a, unsigned int *b, unsigned int *c, double *rd)
{
    unsigned int *dev_a, *dev_b, *dev_c;
    double *dev_rd;
    hipMalloc((void**)&dev_a, N*sizeof(unsigned int)); //alloc gpu memory
    hipMalloc((void**)&dev_b, N*sizeof(unsigned int)); 
    hipMalloc((void**)&dev_c, N*sizeof(unsigned int));
    hipMalloc((void**)&dev_rd, N*sizeof(double));
    hipMemcpy(dev_a, a, N*sizeof(unsigned int), hipMemcpyHostToDevice); // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_b, b, N*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(dev_rd, rd, N*sizeof(double), hipMemcpyHostToDevice);
    
    compare<<<N,1>>>(dev_a, dev_b, dev_rd, dev_c); //call kernel func<<<block, thread>>>
    
    hipMemcpy(c, dev_c, N*sizeof(unsigned int), hipMemcpyDeviceToHost); //copy back the data
    hipMemcpy(rd, dev_rd, N*sizeof(double), hipMemcpyDeviceToHost); //copy back the data
    double sum=0;
    for(std::size_t i=0; i < N; ++i) {
        sum+=rd[i];
    }
    printf("Sum = %lf\n",sum);
    hipFree(dev_a); //memory free
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_rd);
    return 0;
}

int main() {
    unsigned int a[N], b[N], c[N];
    double rd[N];
    for (int i = 0; i < N; ++i) {
        a[i] = rand()%256;
        b[i] = rand()%256;
        rd[i] = ((double)rand()*1.0)/(double)RAND_MAX;
    }
    cpu_compare(a, b, c, rd);
    return 0;
}
