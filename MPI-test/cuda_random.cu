#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <stddef.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/random.h>
#include <time.h>
#include <cuZFP.h>

struct pcg_state_impl {
	uint64_t state;
	uint64_t stream;
};

typedef struct pcg_state_impl pcg_state;

uint32_t pcg32_random(pcg_state* rng)
{
	const uint64_t old = rng->state;
	//Advance internal state
	rng->state = (rng->state) * 0X5851F42D4C957F2DULL;
	// NOLINTNEXTLINE(hicpp-signed-bitwise)
	rng->state += (rng->stream | 1);
	const uint32_t xorshifted = ((old >> 18U) ^ old) >> 27U;
	const uint32_t rot = old >> 59U;
	// NOLINTNEXTLINE(hicpp-signed-bitwise)
	return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
}

int soma_seed_rng(pcg_state* rng, uint64_t seed, uint64_t stream)
{
	rng->stream = stream * 2 + 1;
	rng->state = 0;
	pcg32_random(rng);
	rng->state += seed;
	pcg32_random(rng);
	// Improve quality of first random numbers
	pcg32_random(rng);
	return 0;
}

int main() 
{	
	// initial data in host and device
	size_t size = 1000000;
	size_t byte_size = size * sizeof(double);

	double* a_host = (double*)malloc(byte_size);
	double* b_host = (double*)malloc(byte_size);
	double* c_host = (double*)malloc(byte_size);

	if(a_host == NULL || c_host == NULL) {
		fprintf(stderr, "Failed to allocate host vectors.\n");
		return EXIT_FAILURE;
	}
	uint64_t seed = 0;
	uint64_t stream = 0;
	pcg_state state;

	getrandom(&seed, sizeof(uint64_t), 0);
	soma_seed_rng(&state, seed, stream);

	for(size_t i=0; i<size; ++i) {
		a_host[i] = pcg32_random(&state) / (double)UINT32_MAX;
	}

	double* a_device = NULL;
	double* b_device = NULL;
	double* c_device = NULL;

	hipError_t error_code_a = hipMalloc((void**)&a_device, byte_size);
	//hipError_t error_code_b = hipMalloc((void**)&b_device, byte_size);
	hipError_t error_code_c = hipMalloc((void**)&c_device, byte_size);

	if (error_code_a != hipSuccess || error_code_c != hipSuccess) {
		fprintf(stderr, "Failed to allocate to device vector.\n");
		return EXIT_FAILURE;
	}

	hipError_t error_code_amemc = hipMemcpy(a_device, a_host, byte_size, hipMemcpyHostToDevice);
	if(error_code_amemc != hipSuccess) {
		fprintf(stderr, "Failed to copy memory from host to device");
	}
	
	//compress, decompress
	zfp_stream* zfp;
	bitstream* bstream;
	bitstream* cstream;
	zfp_field *in_field = zfp_field_1d(a_device, zfp_type_double, size);

	int rate = 32;
	int dims = 1;
	zfp = zfp_stream_open(NULL);
	zfp_stream_set_rate(zfp, rate, in_field->type, dims, zfp_false);
	size_t sfpsize;
	size_t buffsize = zfp_stream_maximum_size(zfp, in_field);
	fprintf(stderr, "byte_size = %d\t", byte_size);
	fprintf(stderr, "buffsize = %d\t", buffsize);
	hipError_t error_code_b = hipMalloc((void**)&b_device, buffsize);
	bstream = stream_open(b_device, buffsize);
	zfp_stream_set_bit_stream(zfp, bstream);
	//cuda_compress(zfp, in_field);
	if (zfp_stream_set_execution(zfp, zfp_exec_cuda)) {
		zfpsize = zfp_compress(zfp, in_field);
	} else {
		fprintf(stderr, "Failed to compress");
	}
	stream_close(bstream);
	zfp_field_free(in_field);

	cstream = stream_open(b_device, buffsize);
	zfp_stream_set_bit_stream(zfp, cstream);
	zfp_field *out_field;
	out_field = zfp_field_1d(c_device, zfp_type_double, size);
	cuda_decompress(zfp, out_field);
	if (zfp_stream_set_execution(zfp, zfp_exec_cuda)) {                 
		zfpsize = zfp_decompress(zfp, out_field);
       		fprintf(stderr, "sfpsize = %d\n", zfpsize);		
	} else {
		 fprintf(stderr, "Failed to decompress");
	}
	stream_close(cstream);
	zfp_field_free(out_field);
	zfp_stream_close(zfp);

	hipError_t error_code_cmemc = hipMemcpy(c_host, c_device, byte_size, hipMemcpyDeviceToHost);
	hipError_t error_code_bmemc = hipMemcpy(b_host, b_device, buffsize, hipMemcpyDeviceToHost);
	for(size_t i = 0; i < size; ++i) {
		if(a_host[i] - c_host[i] > pow(10,-6))
			fprintf(stdout, "a_host[%d] = %f, c_host[%d] = %f ", i, a_host[i], i, c_host[i]);
	}
	return EXIT_SUCCESS;
}
